#include "hip/hip_runtime.h"
#include "hipsparse.h"
#include <hip/hip_runtime.h>

#include "utils.h"
#include "time.h"

int main(int argc, char *argv[]) {

  /***********************************************
  *     initialize program's input parameters    *
  ***********************************************/
  double alpha = 1;
  double beta = 0;
  double norm = 0;

  h_vec_t<double> distance_1;
  int num_feat_1 = atoi(argv[2]);
  ReadMatrix(distance_1, argv[1], num_feat_1);
#ifdef ACCELERATE
  std::cout << "CUDA" << std::endl;
  d_vec_t<double> d_distance_1 = distance_1;
#endif

  h_vec_t<double> distance_2;
  int num_feat_2 = atoi(argv[4]);
  ReadMatrix(distance_2, argv[3], num_feat_2);
#ifdef ACCELERATE
  d_vec_t<double> d_distance_2 = distance_2;
#endif
  
  h_vec_t<double> distance_3;
  int num_feat_3 = atoi(argv[6]);
  ReadMatrix(distance_3, argv[5], num_feat_3);
#ifdef ACCELERATE
  d_vec_t<double> d_distance_3 = distance_3;
#endif

  int match_len = atoi(argv[8]);
  h_vec_t<int> matched_feat_1(match_len);
  h_vec_t<int> matched_feat_2(match_len);
  h_vec_t<int> matched_feat_3(match_len);
  ReadMatchedFeatures(matched_feat_1, matched_feat_2, matched_feat_3, argv[7],
                      match_len);
#ifdef ACCELERATE
  d_vec_t<int> d_matched_feat_1 = matched_feat_1;
  d_vec_t<int> d_matched_feat_2 = matched_feat_2;
  d_vec_t<int> d_matched_feat_3 = matched_feat_3;
#endif

  int num_iters = 20;
  if (10 == argc)
    num_iters = atoi(argv[9]);


  /**************************************************
  *            construct affinity matrix            *
  ***************************************************/
  double *distance1 = raw_pointer_cast(distance_1.data());
  double *distance2 = raw_pointer_cast(distance_2.data());
  double *distance3 = raw_pointer_cast(distance_3.data());

  int *h_matched_1 = raw_pointer_cast(matched_feat_1.data());
  int *h_matched_2 = raw_pointer_cast(matched_feat_2.data());
  int *h_matched_3 = raw_pointer_cast(matched_feat_3.data());

  double *affinity = new double[match_len * match_len];
  affinity =
      AffinityInitialMatches(distance1, distance2, distance3, num_feat_1, num_feat_2, num_feat_3, h_matched_1, h_matched_2, h_matched_3, match_len);

#ifdef ACCELERATE
  d_vec_t<double> d_affinity(affinity, affinity + match_len * match_len);
#else
  h_vec_t<double> h_affinity(affinity, affinity + match_len * match_len);
#endif

  /************************************************
  *      convert full matrix to CSR matrix        *
  ************************************************/
  h_vec_t<double> value;
  h_vec_t<int> column;
  h_vec_t<int> row;

  const clock_t begin_time = clock();
  
  CompressMatrix(value, column, row, affinity, match_len, match_len);

  d_vec_t<double> d_value = value;
  d_vec_t<int> d_column = column;
  d_vec_t<int> d_row = row;

  std::cout << "affinity runtime: "
            << float(clock() - begin_time) / CLOCKS_PER_SEC * 1000 << std::endl;
  
  //std::cout << "affinity" << std::endl;
  //std::cout << "values "
  //          << "  "
  //          << "columns"
  //          << "  "
  //          << "rows" << std::endl;
 
 // std::cout << d_value.size() << std::endl;
 // for (int i = 0; i < value.size(); ++i) {
 //   std::cout << value[i] << "    " << column[i] << "      " << std::endl;
 // }
 // std::cout << std::endl;
 // 
 // for (int i = 0; i < row.size(); ++i) {
 //   std::cout << row[i] << "    " << std::endl;
 // }
 // std::cout << std::endl;

  /************************************************
  *           initialize eigen vectors            *
  ************************************************/
  int len_eigen_vec = match_len;
  d_vec_t<double> d_eigen_new(len_eigen_vec);
  fill(d_eigen_new.begin(), d_eigen_new.end(), 0);

  d_vec_t<double> d_eigen_old(len_eigen_vec);
  norm = 1.0 / sqrt(len_eigen_vec);
  fill(d_eigen_old.begin(), d_eigen_old.end(), norm);

  hipsparseHandle_t handle = 0;
  hipsparseMatDescr_t descr = 0;

  ///// create and setup matrix descriptor
  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descr);
  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  /************************************************
  *           computing eigen vector            *
  ************************************************/
  const clock_t begin_time2 = clock();
  
  for (int i = 0; i < num_iters; ++i) {

    hipsparseDcsrmv(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, match_len, match_len,
        d_value.size(), &alpha, descr, raw_pointer_cast(d_value.data()),
        raw_pointer_cast(d_row.data()), raw_pointer_cast(d_column.data()),
        raw_pointer_cast(d_eigen_old.data()), &beta,
        raw_pointer_cast(d_eigen_new.data()));

    double init = 0;
    norm = std::sqrt(transform_reduce(d_eigen_new.begin(), d_eigen_new.end(),
                                      square(), init, thrust::plus<double>()));

    transform(d_eigen_new.begin(), d_eigen_new.end(), d_eigen_old.begin(),
              division(norm));

    fill(d_eigen_new.begin(), d_eigen_new.end(), 0);
  }
  
  std::cout << "Eigen runtime: "
            << float(clock() - begin_time2) / CLOCKS_PER_SEC * 1000 << std::endl;

//  std::cout << "eigen values" << std::endl;
//  for (int i = 0; i < d_eigen_old.size(); i++) {
//    std::cout << "eigen new value = " << d_eigen_new[i] << "  ";
//    std::cout << "eigen old value = " << d_eigen_old[i] << std::endl;
//  }

  hipsparseDestroyMatDescr(descr);
  descr = 0;

  //    destroy handle
  hipsparseDestroy(handle);
  handle = 0;

  return (0);
}

