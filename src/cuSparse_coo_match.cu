#include "hip/hip_runtime.h"
#include "hipsparse.h"
#include <hip/hip_runtime.h>

#include "utils.h"
#include "time.h"

int main(int argc, char *argv[]) {
  /***********************************************
  *     initialize program's input parameters    *
  ***********************************************/
  double alpha = 1;
  double beta = 0;
  double norm = 0;

  hipsparseHandle_t handle = 0;
  hipsparseMatDescr_t descr = 0;
  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descr);

  h_vec_t<double> distance_1;
  int num_feat_1 = atoi(argv[2]);
  ReadMatrix(distance_1, argv[1], num_feat_1);
#ifdef ACCELERATE
  std::cout << "CUDA" << std::endl;
  d_vec_t<double> d_distance_1 = distance_1;
#endif

  h_vec_t<double> distance_2;
  int num_feat_2 = atoi(argv[4]);
  ReadMatrix(distance_2, argv[3], num_feat_2);
#ifdef ACCELERATE
  d_vec_t<double> d_distance_2 = distance_2;
#endif

  h_vec_t<double> distance_3;
  int num_feat_3 = atoi(argv[6]);
  ReadMatrix(distance_3, argv[5], num_feat_3);
#ifdef ACCELERATE
  d_vec_t<double> d_distance_3 = distance_3;
#endif

  int match_len = atoi(argv[8]);
  h_vec_t<int> matched_feat_1(match_len);
  h_vec_t<int> matched_feat_2(match_len);
  h_vec_t<int> matched_feat_3(match_len);
  ReadMatchedFeatures(matched_feat_1, matched_feat_2, matched_feat_3, argv[7],
                      match_len);
#ifdef ACCELERATE
  d_vec_t<int> d_matched_feat_1 = matched_feat_1;
  d_vec_t<int> d_matched_feat_2 = matched_feat_2;
  d_vec_t<int> d_matched_feat_3 = matched_feat_3;
#endif

  int num_iters = 20;
  if (10 == argc)
    num_iters = atoi(argv[9]);

  /**************************************************
  *            construct affinity COO matrix            *
  ***************************************************/
  double *distance1 = raw_pointer_cast(distance_1.data());
  double *distance2 = raw_pointer_cast(distance_2.data());
  double *distance3 = raw_pointer_cast(distance_3.data());

  int *matched_1 = raw_pointer_cast(matched_feat_1.data());
  int *matched_2 = raw_pointer_cast(matched_feat_2.data());
  int *matched_3 = raw_pointer_cast(matched_feat_3.data());

  const clock_t begin_time = clock();
  
  stdvec_tuple_t aff_coo = AffinityInitialmatchesCoo(
      distance1, distance2, distance3, num_feat_1, num_feat_2, num_feat_3,
      matched_1, matched_2, matched_3, match_len);

  //  AffinityInitialmatchesCoo(distance1, distance2, value, column, row,
  //  num_feat_1, num_feat_2, matched_1, matched_2, match_len);
  //
  //  h_vec_t<double> value;
  //  h_vec_t<int> column;
  //  h_vec_t<int> row;

  d_vec_t<double> d_value(get<0>(aff_coo));
  d_vec_t<int> d_column(get<1>(aff_coo));
  d_vec_t<int> d_row(get<2>(aff_coo));

  d_vec_t<int> d_csr_row(match_len + 1);

  hipsparseXcoo2csr(handle, raw_pointer_cast(d_row.data()), d_row.size(),
                   match_len, raw_pointer_cast(d_csr_row.data()),
                   HIPSPARSE_INDEX_BASE_ZERO);

  std::cout << "affinity runtime: "
            << float(clock() - begin_time) / CLOCKS_PER_SEC * 1000 << std::endl;
  
  // for (int i = 0; i < get<0>(aff_coo).size(); ++i) {
  //    std::cout << "val: " << get<0>(aff_coo)[i] << " columns: "<<
  //    get<1>(aff_coo)[i] << " rows: " << get<2>(aff_coo)[i] << std::endl;
  //}
  // std::cout << "affinity" << std::endl;
  // std::cout << "values "
  //          << "  "
  //          << "columns"
  //          << "  "
  //          << "rows" << std::endl;
 // std::cout << d_value.size() << std::endl;
 // for (int i = 0; i < d_value.size(); ++i) {
 //   std::cout << d_value[i] << "     " << d_column[i] << "      " << d_row[i]
 //             << std::endl;
 // }

  /************************************************
  *           initialize eigen vectors            *
  ************************************************/
  int len_eigen_vec = match_len;
  d_vec_t<double> d_eigen_new(len_eigen_vec);
  fill(d_eigen_new.begin(), d_eigen_new.end(), 0);

  d_vec_t<double> d_eigen_old(len_eigen_vec);
  norm = 1.0 / sqrt(len_eigen_vec);
  fill(d_eigen_old.begin(), d_eigen_old.end(), norm);

  hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

  /************************************************
  *           computing eigen vector            *
  ************************************************/
  const clock_t begin_time2 = clock();
  
  for (int i = 0; i < num_iters; ++i) {

    hipsparseDcsrmv(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, match_len, match_len,
        d_value.size(), &alpha, descr, raw_pointer_cast(d_value.data()),
        raw_pointer_cast(d_csr_row.data()), raw_pointer_cast(d_column.data()),
        raw_pointer_cast(d_eigen_old.data()), &beta,
        raw_pointer_cast(d_eigen_new.data()));

    double init = 0;
    norm = std::sqrt(transform_reduce(d_eigen_new.begin(), d_eigen_new.end(),
                                      square(), init, thrust::plus<double>()));

    transform(d_eigen_new.begin(), d_eigen_new.end(), d_eigen_old.begin(),
              division(norm));

    fill(d_eigen_new.begin(), d_eigen_new.end(), 0);
  }

  std::cout << "Eigen runtime: "
            << float(clock() - begin_time2) / CLOCKS_PER_SEC * 1000 << std::endl;

//  std::cout << "eigen values" << std::endl;
//  for (int i = 0; i < d_eigen_old.size(); i++) {
//    std::cout << "eigen new value = " << d_eigen_new[i] << "  ";
//    std::cout << "eigen old value = " << d_eigen_old[i] << std::endl;
//  }

  hipsparseDestroyMatDescr(descr);
  descr = 0;

  //    destroy handle
  hipsparseDestroy(handle);
  handle = 0;

  return (0);
}

